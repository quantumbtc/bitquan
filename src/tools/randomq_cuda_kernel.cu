#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>

// CUDA version of RandomQ constants
__constant__ uint64_t RANDOMQ_CONSTANTS[25] = {
    0x6a09e667f3bcc908ULL, 0xbb67ae8584caa73bULL, 0x3c6ef372fe94f82bULL,
    0xa54ff53a5f1d36f1ULL, 0x510e527fade682d1ULL, 0x9b05688c2b3e6c1fULL,
    0x1f83d9abfb41bd6bULL, 0x5be0cd19137e2179ULL, 0x428a2f98d728ae22ULL,
    0x7137449123ef65cdULL, 0xb5c0fbcfec4d3b2fULL, 0xe9b5dba58189dbbcULL,
    0x3956c25bf348b538ULL, 0x59f111f1b605d019ULL, 0x923f82a4af194f9bULL,
    0xab1c5ed5da6d8118ULL, 0xd807aa98a3030242ULL, 0x12835b0145706fbeULL,
    0x243185be4ee4b28cULL, 0x550c7dc3d5ffb4e2ULL, 0x72be5d74f27b896fULL,
    0x80deb1fe3b1696b1ULL, 0x9bdc06a725c71235ULL, 0xc19bf174cf692694ULL,
    0xe49b69c19ef14ad2ULL
};

// SHA256 constants
__constant__ uint32_t K256[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffaU, 0xa4506cebU, 0xbef9a3f7U, 0xc67178f2U
};

// CUDA device functions
__device__ inline uint32_t rotr32(uint32_t x, uint32_t r) { 
    return (x >> r) | (x << (32 - r)); 
}

__device__ inline uint32_t Ch(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ ((~x) & z);
}

__device__ inline uint32_t Maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (x & z) ^ (y & z);
}

__device__ inline uint32_t Sigma0(uint32_t x) {
    return rotr32(x, 2) ^ rotr32(x, 13) ^ rotr32(x, 22);
}

__device__ inline uint32_t Sigma1(uint32_t x) {
    return rotr32(x, 6) ^ rotr32(x, 11) ^ rotr32(x, 25);
}

__device__ inline uint32_t sigma0(uint32_t x) {
    return rotr32(x, 7) ^ rotr32(x, 18) ^ (x >> 3);
}

__device__ inline uint32_t sigma1(uint32_t x) {
    return rotr32(x, 17) ^ rotr32(x, 19) ^ (x >> 10);
}

// SHA256 implementation for CUDA
__device__ void sha256_cuda(const uint8_t* msg, uint32_t len, uint8_t* out32) {
    uint8_t tmp[256];
    
    // Initialize with zeros
    for (int i = 0; i < 256; ++i) tmp[i] = 0;
    
    // Copy message
    for (uint32_t i = 0; i < len; ++i) tmp[i] = msg[i];
    
    // Add padding
    tmp[len] = 0x80;
    uint64_t bitlen = ((uint64_t)len) * 8ULL;
    
    // Calculate number of blocks
    uint32_t blocks = (len + 9 + 63) / 64;
    uint32_t last_index = blocks * 64;
    
    // Set length in last 8 bytes (big-endian)
    tmp[last_index - 8] = (uint8_t)((bitlen >> 56) & 0xFF);
    tmp[last_index - 7] = (uint8_t)((bitlen >> 48) & 0xFF);
    tmp[last_index - 6] = (uint8_t)((bitlen >> 40) & 0xFF);
    tmp[last_index - 5] = (uint8_t)((bitlen >> 32) & 0xFF);
    tmp[last_index - 4] = (uint8_t)((bitlen >> 24) & 0xFF);
    tmp[last_index - 3] = (uint8_t)((bitlen >> 16) & 0xFF);
    tmp[last_index - 2] = (uint8_t)((bitlen >> 8) & 0xFF);
    tmp[last_index - 1] = (uint8_t)(bitlen & 0xFF);
    
    // Initialize hash values
    uint32_t h0 = 0x6a09e667, h1 = 0xbb67ae85, h2 = 0x3c6ef372, h3 = 0xa54ff53a;
    uint32_t h4 = 0x510e527f, h5 = 0x9b05688c, h6 = 0x1f83d9ab, h7 = 0x5be0cd19;
    
    uint32_t W[64];
    
    // Process blocks
    for (uint32_t block = 0; block < blocks; ++block) {
        // Prepare W[0..15]
        for (int t = 0; t < 16; ++t) {
            uint32_t base = block * 64 + t * 4;
            uint32_t w = ((uint32_t)tmp[base + 0] << 24) |
                         ((uint32_t)tmp[base + 1] << 16) |
                         ((uint32_t)tmp[base + 2] << 8) |
                         ((uint32_t)tmp[base + 3] << 0);
            W[t] = w;
        }
        
        // Extend W[16..63]
        for (int t = 16; t < 64; ++t) {
            uint32_t s0 = sigma0(W[t-15]);
            uint32_t s1 = sigma1(W[t-2]);
            W[t] = W[t-16] + s0 + W[t-7] + s1;
        }
        
        // Initialize working variables
        uint32_t a = h0, b = h1, c = h2, d = h3, e = h4, f = h5, g = h6, h = h7;
        
        // Main loop
        for (int t = 0; t < 64; ++t) {
            uint32_t T1 = h + Sigma1(e) + Ch(e, f, g) + K256[t] + W[t];
            uint32_t T2 = Sigma0(a) + Maj(a, b, c);
            h = g; g = f; f = e; e = d + T1; d = c; c = b; b = a; a = T1 + T2;
        }
        
        // Add to hash values
        h0 += a; h1 += b; h2 += c; h3 += d; h4 += e; h5 += f; h6 += g; h7 += h;
    }
    
    // Output hash (big-endian)
    out32[0] = (uint8_t)((h0 >> 24) & 0xFF);
    out32[1] = (uint8_t)((h0 >> 16) & 0xFF);
    out32[2] = (uint8_t)((h0 >> 8) & 0xFF);
    out32[3] = (uint8_t)((h0 >> 0) & 0xFF);
    
    out32[4] = (uint8_t)((h1 >> 24) & 0xFF);
    out32[5] = (uint8_t)((h1 >> 16) & 0xFF);
    out32[6] = (uint8_t)((h1 >> 8) & 0xFF);
    out32[7] = (uint8_t)((h1 >> 0) & 0xFF);
    
    out32[8] = (uint8_t)((h2 >> 24) & 0xFF);
    out32[9] = (uint8_t)((h2 >> 16) & 0xFF);
    out32[10] = (uint8_t)((h2 >> 8) & 0xFF);
    out32[11] = (uint8_t)((h2 >> 0) & 0xFF);
    
    out32[12] = (uint8_t)((h3 >> 24) & 0xFF);
    out32[13] = (uint8_t)((h3 >> 16) & 0xFF);
    out32[14] = (uint8_t)((h3 >> 8) & 0xFF);
    out32[15] = (uint8_t)((h3 >> 0) & 0xFF);
    
    out32[16] = (uint8_t)((h4 >> 24) & 0xFF);
    out32[17] = (uint8_t)((h4 >> 16) & 0xFF);
    out32[18] = (uint8_t)((h4 >> 8) & 0xFF);
    out32[19] = (uint8_t)((h4 >> 0) & 0xFF);
    
    out32[20] = (uint8_t)((h5 >> 24) & 0xFF);
    out32[21] = (uint8_t)((h5 >> 16) & 0xFF);
    out32[22] = (uint8_t)((h5 >> 8) & 0xFF);
    out32[23] = (uint8_t)((h5 >> 0) & 0xFF);
    
    out32[24] = (uint8_t)((h6 >> 24) & 0xFF);
    out32[25] = (uint8_t)((h6 >> 16) & 0xFF);
    out32[26] = (uint8_t)((h6 >> 8) & 0xFF);
    out32[27] = (uint8_t)((h6 >> 0) & 0xFF);
    
    out32[28] = (uint8_t)((h7 >> 24) & 0xFF);
    out32[29] = (uint8_t)((h7 >> 16) & 0xFF);
    out32[30] = (uint8_t)((h7 >> 8) & 0xFF);
    out32[31] = (uint8_t)((h7 >> 0) & 0xFF);
}

// RandomQ context structure
struct CRANDOMQ_CTX {
    uint64_t state[25];
    uint64_t nonce;
    uint64_t rounds;
};

__device__ inline void CRandomQ_Reset(CRANDOMQ_CTX* ctx) {
    for (int i = 0; i < 25; ++i) ctx->state[i] = RANDOMQ_CONSTANTS[i];
    ctx->nonce = 0ULL;
    ctx->rounds = 8192ULL;
}

__device__ inline void CRandomQ_SetRounds(CRANDOMQ_CTX* ctx, uint64_t rounds) {
    ctx->rounds = rounds;
}

__device__ inline void CRandomQ_SetNonce(CRANDOMQ_CTX* ctx, uint64_t nonce) {
    ctx->nonce = nonce;
}

__device__ inline void CRandomQ_RandomQRound(CRANDOMQ_CTX* ctx) {
    // Rotate and mix
    for (int i = 0; i < 25; ++i) {
        uint64_t v = ctx->state[i];
        uint64_t rotated = (v << 13) | (v >> (64 - 13));
        uint64_t next = ctx->state[(i + 1) % 25];
        ctx->state[i] = rotated ^ next ^ (v + next);
        // Add constant
        ctx->state[i] += RANDOMQ_CONSTANTS[i];
    }
    
    // Additional mixing
    for (int i = 0; i < 25; i += 2) {
        uint64_t temp = ctx->state[i];
        ctx->state[i] = ctx->state[i] ^ ctx->state[(i + 1) % 25];
        ctx->state[(i + 1) % 25] = ctx->state[(i + 1) % 25] ^ temp;
    }
}

__device__ inline void CRandomQ_Write(CRANDOMQ_CTX* ctx, const uint8_t* input, uint32_t len) {
    if (len == 0) return;
    uint32_t offset = 0;
    while (offset < len) {
        uint32_t chunk_size = (len - offset) > 64 ? 64 : (len - offset);
        // Mix input chunk into state (up to 8 uint64 words)
        uint32_t words = chunk_size / 8;
        if (words > 8) words = 8;
        for (uint32_t i = 0; i < words; ++i) {
            uint64_t chunk = 0ULL;
            for (uint32_t j = 0; j < 8; ++j) {
                uint32_t idx = offset + i * 8 + j;
                uint8_t b = (idx < len) ? input[idx] : 0;
                chunk |= ((uint64_t)b) << (j * 8); // little-endian assembly
            }
            ctx->state[i] ^= chunk;
        }
        // Run one round
        CRandomQ_RandomQRound(ctx);
        offset += chunk_size;
    }
}

__device__ inline void CRandomQ_StateToHash(CRANDOMQ_CTX* ctx, uint8_t out[32]) {
    // Use sha256 on the 25*8 = 200 state bytes; bytes are written little-endian
    uint8_t tmp[200];
    for (int i = 0; i < 25; ++i) {
        uint64_t v = ctx->state[i];
        for (int j = 0; j < 8; ++j) {
            tmp[i * 8 + j] = (uint8_t)((v >> (j * 8)) & 0xFF); // little-endian bytes
        }
    }
    // Compute sha256 over tmp (200 bytes)
    sha256_cuda(tmp, 200, out);
}

__device__ inline void CRandomQ_Finalize(CRANDOMQ_CTX* ctx, uint8_t out[32]) {
    // Mix nonce
    ctx->state[0] ^= ctx->nonce;
    // Run rounds
    for (uint64_t i = 0; i < ctx->rounds; ++i) {
        CRandomQ_RandomQRound(ctx);
    }
    // Convert state to hash via SHA256
    CRandomQ_StateToHash(ctx, out);
}

// Main CUDA kernel for RandomQ mining
__global__ void randomq_mining_kernel(
    uint8_t* header,        // 80 bytes block header
    uint32_t* nonce_base,   // base nonce
    uint8_t* target,        // 32 bytes target (little-endian)
    uint32_t* found_flag,   // found flag
    uint32_t* found_nonce,  // found nonce
    uint8_t* result_hash    // 32 bytes result hash
) {
    uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t current_nonce = *nonce_base + gid;
    
    // Build local header and inject nonce
    uint8_t local_header[80];
    for (int i = 0; i < 80; ++i) local_header[i] = header[i];
    
    // Nonce in header bytes 76..79 (little-endian)
    local_header[76] = (uint8_t)((current_nonce) & 0xFF);
    local_header[77] = (uint8_t)((current_nonce >> 8) & 0xFF);
    local_header[78] = (uint8_t)((current_nonce >> 16) & 0xFF);
    local_header[79] = (uint8_t)((current_nonce >> 24) & 0xFF);
    
    // Step 1: First SHA256(header)
    uint8_t first_sha[32];
    sha256_cuda(local_header, 80, first_sha);
    
    // Step 2: CRandomQ processing
    CRANDOMQ_CTX ctx;
    CRandomQ_Reset(&ctx);
    CRandomQ_SetRounds(&ctx, 8192ULL);
    CRandomQ_SetNonce(&ctx, (uint64_t)current_nonce);
    CRandomQ_Write(&ctx, first_sha, 32);
    uint8_t randomq_out[32];
    CRandomQ_Finalize(&ctx, randomq_out);
    
    // Step 3: Final SHA256(randomq_out)
    uint8_t final32[32];
    sha256_cuda(randomq_out, 32, final32);
    
    // Convert final32 to little-endian for comparison
    uint8_t final_le[32];
    for (int i = 0; i < 32; ++i) {
        final_le[i] = final32[31 - i];
    }
    
    // Compare final_le (little-endian) with target (little-endian) from MSB to LSB
    bool meets_target = true;
    for (int i = 31; i >= 0; --i) {
        uint8_t hb = final_le[i];
        uint8_t tb = target[i];
        if (hb > tb) {
            meets_target = false;
            break;
        } else if (hb < tb) {
            meets_target = true;
            break;
        }
        // If equal, continue to next byte
    }
    
    if (meets_target) {
        // Atomic set found_flag
        uint32_t old = atomicCAS(found_flag, 0, 1);
        if (old == 0) {
            *found_nonce = current_nonce;
            for (int i = 0; i < 32; ++i) result_hash[i] = final_le[i];
        }
    }
}

// Host function to launch the kernel
extern "C" void launch_randomq_kernel(
    uint8_t* d_header,
    uint32_t* d_nonce_base,
    uint8_t* d_target,
    uint32_t* d_found_flag,
    uint32_t* d_found_nonce,
    uint8_t* d_result_hash,
    uint32_t grid_size,
    uint32_t block_size,
    hipStream_t stream
) {
    randomq_mining_kernel<<<grid_size, block_size, 0, stream>>>(
        d_header, d_nonce_base, d_target, d_found_flag, d_found_nonce, d_result_hash
    );
}

// Device query function
extern "C" void cuda_device_query() {
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    
    printf("CUDA Device Query:\n");
    printf("Found %d CUDA device(s)\n", device_count);
    
    for (int i = 0; i < device_count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Global Memory: %.1f MB\n", (double)prop.totalGlobalMem / (1024 * 1024));
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Grid Size: %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("  Max Block Size: %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Warp Size: %d\n", prop.warpSize);
        printf("  Memory Clock Rate: %.1f MHz\n", prop.memoryClockRate * 1e-3);
        printf("  Memory Bus Width: %d bits\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth: %.1f GB/s\n", 
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("\n");
    }
}
